
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <hipblas.h>
#include <time.h>

//define N
//define threads/block


//FILE IO RELATED
//max number of lines in the training dataset
#define MAX_ROWS_TRAINING 17012
// max number of columns/features in the training dataset
#define MAX_COLUMNS_TRAINING 26
// max number of rows in the testing dataset
#define MAX_ROWS_TESTING 4252
// max number of columns in the testing data
#define MAX_COLUMNS_TESTING 26
//max number of characters/line
#define MAX_CHAR 300

bool LoadCSV(float** data, char* filename, int pRows, int pCols) {
    //assumed file is in same folder, also rename file here
    FILE *file;
    file = fopen(filename, "r+");
    if(!file) {
        printf("Can't open file \n");
        return false;
    }
    // unparsed data straight from file
    char unparsed_data[pRows+10][pCols+10];
    char copying[pRows+10];

    //keep track which row we are on
    int ltracker=0;
    while(fgets(copying, sizeof(copying)-10, file) != 0){
        //copying the line from temp to our array
        strncpy(unparsed_data[ltracker], copying, MAX_CHAR);
        ltracker++;
    }
    //closing the file
    fclose(file);
    char* col_val;
    const char deli[2]=","; // delimiter
    // parses each value in each column per row
    for (int row = 0; row < pRows; row++){
        col_val = strtok(unparsed_data[row], deli);
        for(int col  = 0; col < pCols; col++) {
            col_val = strtok(NULL, deli);
            if(col_val != NULL) {
                data[row][col] = atof(col_val);
            }
        }
    }
    return true;
}

//on the cpu
int main(void){
    // array that holds all converted training data
    float **training_data = (float **) malloc(MAX_ROWS_TRAINING * sizeof(float *));
    for(int i = 0; i < MAX_ROWS_TRAINING; i++) {
        training_data[i] = (float *) malloc(MAX_COLUMNS_TRAINING * sizeof(float));
    }
    if(LoadCSV(training_data, "training_data.csv", MAX_ROWS_TRAINING, MAX_COLUMNS_TRAINING)) {
        for(int i = 0; i < MAX_ROWS_TRAINING; i++) {
            printf("Row %i: ", i );
            for(int j = 0; j < MAX_COLUMNS_TRAINING; j++) {
                printf("%f, ", training_data[i][j]);
            }
            printf("\n");
            break;
        }
    }
    //data parsing completed

    //TODO: store the lines in the file as arrays
    //unparsed_data[0] = [colum1, colum2, column3, .. ,columnN]

    //things on host
    //things to device
    //used to set size of components


    //alloc space for device, copies of above

    //alloc space for host and setup input values

    //copy inputs to device

    //launch on gpu

    //copy result to host

    //cleanup all the frees

    return 0;
}
