#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

/* Test cuda file that writes to an array all 0's. */

#define RESULT_SIZE 12

__constant__ int features = 5;

__global__ void SetZero(int* result) {
    int index = blockIdx.x;
    result[index] = 0;
}

__global__ void mult(int* results, int* data, int* vec) {
    int index = blockIdx.x;
    int result_val = 0;
    for(int i = 0; i < features; i++) {
        result_val += vec[i] * data[(index * features) + i];
    }
    results[index] = result_val;
}

int main() {
    // arr exists on the CPU/host
    int rows = 6;
    int* vec = (int*) malloc(sizeof(int) * features * 1);
    for(int i = 0; i < features; i++) {
        vec[i] = i;
    }
    int* data = (int*) malloc(sizeof(int) * features * rows);
    for(int i = 0; i < features * rows; i++) {
        data[i] = 1;
    }

    int* result = (int*) malloc(sizeof(int)* features);
    // copy vector and data to gpu
    int* gpu_vec;
    int* gpu_data;
    int* gpu_result;
    hipMalloc((void**)&gpu_vec, sizeof(int) * features);
	hipMalloc((void**)&gpu_data, sizeof(int) * features * rows);
	hipMalloc((void**)&gpu_result, sizeof(int) * features);

    hipMemcpy(gpu_vec, vec, sizeof(int) * features, hipMemcpyHostToDevice);
    hipMemcpy(gpu_data, data, sizeof(int) * features * rows hipMemcpyHostToDevice);

    dim3 grid(rows);
    mult<<<rows, 1>>>(gpu_result, gpu_data, gpu_vec);
    // copy back to device
    hipMemcpy(result, gpu_result, sizeof(int) * features, hipMemcpyDeviceToHost);
    // check all of our result
    for(int i = 0; i < features; i++) {
        printf("%d \n", result[i]);
    }
}
